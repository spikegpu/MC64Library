#include <iostream>
#include <cmath>

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>

using std::cout;
using std::cerr;
using std::endl;

void old_transform() {
	const int ARRAY_SIZE = 1000;

	double *hA, *dA;
	hA = (double *) malloc(ARRAY_SIZE * sizeof(double));
	hipMalloc(&dA, ARRAY_SIZE * sizeof(double));

	for (int i = 0; i < ARRAY_SIZE; i++)
		hA[i] = 1.0 * (i + 1);

	hipMemcpy(dA, hA, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);
	/* FIXME: it is not correct to use
	thrust::transform(thrust::cuda::par, dA, dA + ARRAY_SIZE, dA, thrust::negate<double>());
	*/
	{
		thrust::device_ptr<double> A_begin(dA);
		thrust::device_ptr<double> A_end(dA + ARRAY_SIZE);
		thrust::transform(thrust::cuda::par, A_begin, A_end, A_begin, thrust::negate<double>());
	}
	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);

	bool correct = true;
	for (int i = 0; i < ARRAY_SIZE; i++)
		if (hA[i] != - 1.0 * (i + 1)) {
			correct = false;
			break;
		}

	if (correct)
		cout << "Old transform: correct" << endl;
	else
		cout << "Old transform: INCORRECT" << endl;

	hipFree(dA);
	free(hA);
}

void new_transform() {
	const int ARRAY_SIZE = 1000;

	double *mA;
	hipMallocManaged(&mA, ARRAY_SIZE * sizeof(double));

	for (int i = 0; i < ARRAY_SIZE; i++)
		mA[i] = 1.0 * (i + 1);

	/* FIXME: it is not correct to use
	thrust::transform(thrust::cuda::par, mA, mA + ARRAY_SIZE, mA, thrust::negate<double>());
	*/
	{
		thrust::device_ptr<double> A_begin(mA);
		thrust::device_ptr<double> A_end(mA + ARRAY_SIZE);
		thrust::transform(thrust::cuda::par, A_begin, A_end, A_begin, thrust::negate<double>());
	}
	hipDeviceSynchronize();

	bool correct = true;
	for (int i = 0; i < ARRAY_SIZE; i++)
		if (mA[i] != - 1.0 * (i + 1)) {
			correct = false;
			break;
		}

	if (correct)
		cout << "New transform: correct" << endl;
	else
		cout << "New transform: INCORRECT" << endl;

	hipFree(mA);
}

void old_transform_if() {
	const int ARRAY_SIZE = 1000;

	double *hA, *dA, *hB, *dB;
	int *h_stencil, *d_stencil;

	hA = (double *) malloc(ARRAY_SIZE * sizeof(double));
	hipMalloc(&dA, ARRAY_SIZE * sizeof(double));
	hB = (double *) malloc(ARRAY_SIZE * sizeof(double));
	hipMalloc(&dB, ARRAY_SIZE * sizeof(double));
	h_stencil = (int *) malloc(ARRAY_SIZE * sizeof(int));
	hipMalloc(&d_stencil, ARRAY_SIZE * sizeof(int));

	for (int i = 0; i < ARRAY_SIZE; i++)
		hB[i] = hA[i] = 1.0 * (i + 1);

	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i < (ARRAY_SIZE >> 1))
			h_stencil[i] = 1;
		else
			h_stencil[i] = 0;
	}

	hipMemcpy(dA, hA, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_stencil, h_stencil, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);
	/* FIXME: it is not correct to use
	thrust::transform_if(thrust::cuda::par, dA, dA + ARRAY_SIZE, dB, d_stencil, dA, thrust::plus<double>(), thrust::identity<int>());
	*/
	{
		thrust::device_ptr<double> A_begin(dA);
		thrust::device_ptr<double> A_end(dA + ARRAY_SIZE);
		thrust::device_ptr<double> B_begin(dB);
		thrust::device_ptr<int>    stencil_begin(d_stencil);
		thrust::transform_if(thrust::cuda::par, A_begin, A_end, B_begin, stencil_begin, A_begin, thrust::plus<double>(), thrust::identity<int>());
	}
	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);

	bool correct = true;
	for (int i = 0; i < (ARRAY_SIZE >> 1); i++)
		if (hA[i] != 2.0 * (i + 1)) {
			correct = false;
			break;
		}

	if (correct) {
		for (int i = (ARRAY_SIZE >> 1); i < ARRAY_SIZE; i++)
			if (hA[i] != 1.0 * (i + 1)) {
				correct = false;
				break;
			}
	}

	if (correct)
		cout << "Old transform_if: correct" << endl;
	else
		cout << "Old transform_if: INCORRECT" << endl;

	hipFree(dA);
	hipFree(dB);
	hipFree(d_stencil);
	free(hA);
	free(hB);
	free(h_stencil);
}

void new_transform_if() {
	const int ARRAY_SIZE = 1000;

	double *mA, *mB;
	int *m_stencil;

	hipMallocManaged(&mA, ARRAY_SIZE * sizeof(double));
	hipMallocManaged(&mB, ARRAY_SIZE * sizeof(double));
	hipMallocManaged(&m_stencil, ARRAY_SIZE * sizeof(int));

	for (int i = 0; i < ARRAY_SIZE; i++)
		mB[i] = mA[i] = 1.0 * (i + 1);

	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i < (ARRAY_SIZE >> 1))
			m_stencil[i] = 1;
		else
			m_stencil[i] = 0;
	}

	/* FIXME: it is not correct to use 
	thrust::transform_if(thrust::cuda::par, mA, mA + ARRAY_SIZE, mB, m_stencil, mA, thrust::plus<double>(), thrust::identity<int>());
	*/

	{ 
		thrust::device_ptr<double> A_begin(mA);
		thrust::device_ptr<double> A_end(mA + ARRAY_SIZE);
		thrust::device_ptr<double> B_begin(mB);
		thrust::device_ptr<int>    stencil_begin(m_stencil);
		thrust::transform_if(thrust::cuda::par, A_begin, A_end, B_begin, stencil_begin, A_begin, thrust::plus<double>(), thrust::identity<int>());
	}
	hipDeviceSynchronize();

	bool correct = true;
	for (int i = 0; i < (ARRAY_SIZE >> 1); i++)
		if (mA[i] != 2.0 * (i + 1)) {
			correct = false;
			break;
		}

	if (correct) {
		for (int i = (ARRAY_SIZE >> 1); i < ARRAY_SIZE; i++)
			if (mA[i] != 1.0 * (i + 1)) {
				correct = false;
				break;
			}
	}

	if (correct)
		cout << "New transform_if: correct" << endl;
	else
		cout << "New transform_if: INCORRECT" << endl;

	hipFree(mA);
	hipFree(mB);
	hipFree(m_stencil);
}

int main(int argc, char **argv) 
{
	old_transform();
	new_transform();
	old_transform_if();
	new_transform_if();
	return 0;
}
